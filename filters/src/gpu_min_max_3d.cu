#include "hip/hip_runtime.h"
//
// Created by konrad on 11/8/17.
//

#include "pcl/filters/boost.h"
#include "../../../../../../usr/local/cuda-8.0/include/hip/driver_types.h"
#include <pcl/filters/filter.h>
#include <pcl/filters/gpu_min_max_3d.h>
#include <Eigen/Core>
#define HAVE_CUDA
#ifdef HAVE_CUDA
#include <hip/hip_runtime.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

#endif

namespace pcl {
    namespace filters {
        namespace gpu {
#ifndef HAVE_CUDA

            constexpr bool getMinMax3D(const pcl::PCLPointCloud2ConstPtr &cloud, int x_idx, int y_idx, int z_idx,
                                       Eigen::Vector4f &min_pt, Eigen::Vector4f &max_pt) {
                return false;
            }

            constexpr bool getMinMax3D(const pcl::PCLPointCloud2ConstPtr &cloud, int x_idx, int y_idx, int z_idx,
                                       const std::string &distance_field_name, float min_distance, float max_distance,
                                       Eigen::Vector4f &min_pt, Eigen::Vector4f &max_pt, bool limit_negative = false) {
                return false;
            }

#else
            template<typename T>
            struct convert_indefinite_to_min_value : public thrust::unary_function<T,T>
            {
                __host__ __device__ T operator()(const T &x) const
                {
                    if (!isfinite (x[0]) ||
                               !isfinite (x[1]) ||
                               !isfinite (x[2])) {
                        T result;
                        result.setConstant (-FLT_MAX);
                        return result;
                    } else {
                        return x;
                    }
                }
            };

            template<typename T>
            struct convert_indefinite_to_max_value : public thrust::unary_function<T,T>
            {
                __host__ __device__ T operator()(const T &x) const
                {
                    if (!isfinite (x[0]) ||
                        !isfinite (x[1]) ||
                        !isfinite (x[2])) {
                        T result;
                        result.setConstant (FLT_MAX);
                        return result;
                    } else {
                        return x;
                    }
                }
            };

            template<typename T>
            class convert_indefinite_to_min_value_with_limits : public thrust::unary_function<T,T>
            {
                float max_limit_;
                float min_limit_;
                bool limit_negative_;
                int distance_index_;
                const int x_index_;
                const int y_index_;
                const int z_index_;

            public:
                convert_indefinite_to_min_value_with_limits(float min_limit, float max_limit, bool limit_negative, int distance_index, int x_index, int y_index, int z_index) : thrust::unary_function<T, T>(), min_limit_(min_limit), max_limit_(max_limit), limit_negative_(limit_negative), distance_index_(distance_index), x_index_(x_index), y_index_(y_index), z_index_(z_index)
                {
                }

                __host__ __device__ T operator()(const T &x) const
                {
                    // Get the distance value
                    float distance_value = 0;
                    if(distance_index_ == x_index_) {
                        distance_value = x[x_index_];
                    } else if (distance_index_ == y_index_) {
                        distance_value = x[y_index_];
                    } else {
                        distance_value = x[z_index_];
                    }
                    if (limit_negative_)
                    {
                        // Use a threshold for cutting out points which inside the interval
                        if ((distance_value < max_limit_) && (distance_value > min_limit_))
                        {
                            T result;
                            result.setConstant (-FLT_MAX);
                            return result;
                        }
                    }
                    else
                    {
                        // Use a threshold for cutting out points which are too close/far away
                        if ((distance_value > max_limit_) || (distance_value < min_limit_))
                        {
                            T result;
                            result.setConstant (-FLT_MAX);
                            return result;
                        }
                    }


                    if (!isfinite (x[0]) ||
                        !isfinite (x[1]) ||
                        !isfinite (x[2])) {
                        T result;
                        result.setConstant (-FLT_MAX);
                        return result;
                    } else {
                        return x;
                    }
                }
            };

            template<typename T>
            class convert_indefinite_to_max_value_with_limits : public thrust::unary_function<T,T>
            {
                float max_limit_;
                float min_limit_;
                bool limit_negative_;
                int distance_index_;
                int x_index_;
                int y_index_;
                int z_index_;

            public:
                convert_indefinite_to_max_value_with_limits(float min_limit, float max_limit, bool limit_negative, int distance_index, int x_index, int y_index, int z_index) : thrust::unary_function<T, T>(), min_limit_(min_limit), max_limit_(max_limit), limit_negative_(limit_negative), distance_index_(distance_index), x_index_(x_index), y_index_(y_index), z_index_(z_index) {
                }
                __host__ __device__ T operator()(const T &x) const
                {
                    // Get the distance value
                    float distance_value = 0;
                    if(distance_index_ == x_index_) {
                        distance_value = x[x_index_];
                    } else if (distance_index_ == y_index_) {
                        distance_value = x[y_index_];
                    } else {
                        distance_value = x[z_index_];
                    }
                    if (limit_negative_)
                    {
                        // Use a threshold for cutting out points which inside the interval
                        if ((distance_value < max_limit_) && (distance_value > min_limit_))
                        {
                            T result;
                            result.setConstant (FLT_MAX);
                            return result;
                        }
                    }
                    else
                    {
                        // Use a threshold for cutting out points which are too close/far away
                        if ((distance_value > max_limit_) || (distance_value < min_limit_))
                        {
                            T result;
                            result.setConstant (FLT_MAX);
                            return result;
                        }
                    }


                    if (!isfinite (x[0]) ||
                        !isfinite (x[1]) ||
                        !isfinite (x[2])) {
                        T result;
                        result.setConstant (FLT_MAX);
                        return result;
                    } else {
                        return x;
                    }
                }
            };

            template<typename T>
            struct compute_minimum_of_bounding_box : public thrust::binary_function<T,T,T>
            {
                __host__ __device__ T operator()(const T &x, const T&y) const
                {
                    T result = x;
                    result.min(y);
                    return result;
                }
            };

            template<typename T>
            struct compute_maximum_of_bounding_box : public thrust::binary_function<T,T,T>
            {
                __host__ __device__ T operator()(const T &x, const T&y) const
                {
                    T result = x;
                    result.max(y);
                    return result;
                }
            };

            __global__ void convertToDeviceVector(unsigned char * data, int nr_points, int xyz_offset0, int xyz_offset1, int xyz_offset2, int point_step, Eigen::Array4f* all_distance_points) {
                int point = blockIdx.x * blockDim.x + threadIdx.x;
                if (point >= nr_points) {
                    return;
                }

                int point_address = point * point_step;
                Eigen::Array4f pt = Eigen::Array4f::Zero ();


                memcpy (&pt[0], &data[point_address + xyz_offset0], sizeof (float));
                memcpy (&pt[1], &data[point_address + xyz_offset1], sizeof (float));
                memcpy (&pt[2], &data[point_address + xyz_offset2], sizeof (float));

                all_distance_points[point] = pt;
            }


            __host__ bool getMinMax3D (const pcl::PCLPointCloud2ConstPtr &cloud, int x_idx, int y_idx, int z_idx,
                              Eigen::Vector4f &min_pt, Eigen::Vector4f &max_pt) {
                // @todo fix this
                if (cloud->fields[x_idx].datatype != pcl::PCLPointField::FLOAT32 ||
                    cloud->fields[y_idx].datatype != pcl::PCLPointField::FLOAT32 ||
                    cloud->fields[z_idx].datatype != pcl::PCLPointField::FLOAT32)
                {
                    PCL_ERROR ("[pcl::getMinMax3D] XYZ dimensions are not float type!\n");
                    return false;
                }

                Eigen::Array4f min_p, max_p;
                min_p.setConstant (FLT_MAX);
                max_p.setConstant (-FLT_MAX);

                size_t nr_points = cloud->width * cloud->height;

                Eigen::Array4f pt = Eigen::Array4f::Zero ();
                hipInit(0);
                hipCtx_t context;
                hipDevice_t device;
                hipDeviceGet ( &device, 0 );
                hipCtxCreate(&context, hipDeviceScheduleBlockingSync | hipDeviceMapHost,device);
                hipCtxPushCurrent(context);
                hipStream_t stream;
                hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
                thrust::device_vector<Eigen::Array4f> all_distance_points(nr_points);

                uint8_t * gpu_data_cloud;
                hipMalloc(&gpu_data_cloud, sizeof(cloud->data[0]) * cloud->data.size());
                hipMemcpy(gpu_data_cloud, cloud->data.data(), sizeof(cloud->data[0]) * cloud->data.size(), hipMemcpyHostToDevice );
                //hipHostRegister(reinterpret_cast<void *>(const_cast<uint8_t *>(cloud->data.data())), sizeof(cloud->data[0]) * cloud->data.size(), hipHostRegisterPortable | hipHostRegisterMapped);
                //hipHostGetDevicePointer();

                //hipDeviceptr_t device_ptr;
                //hipHostGetDevicePointer(&device_ptr, reinterpret_cast<void *>(const_cast<uint8_t *>(cloud->data.data())), 0);
                //cuMemcpy();
                dim3 threadsPerBlock(1024, 1);
                dim3 numBlocks(static_cast<int> (std::ceil(static_cast<float>(cloud->data.size()) / threadsPerBlock.x)),
                               1);

                //kernel launch

                convertToDeviceVector<<<threadsPerBlock, numBlocks, 0, stream>>>((unsigned char *)gpu_data_cloud, nr_points, cloud->fields[x_idx].offset, cloud->fields[y_idx].offset, cloud->fields[z_idx].offset, cloud->point_step, thrust::raw_pointer_cast(&all_distance_points[0]));

                convert_indefinite_to_max_value<Eigen::Array4f> f1;
                compute_minimum_of_bounding_box<Eigen::Array4f> f2;
                convert_indefinite_to_min_value<Eigen::Array4f> g1;
                compute_maximum_of_bounding_box<Eigen::Array4f> g2;

                auto min_value = thrust::transform_reduce(thrust::cuda::par.on(stream), all_distance_points.begin(), all_distance_points.end(), f1, min_p, f2);
                auto max_value = thrust::transform_reduce(thrust::cuda::par.on(stream), all_distance_points.begin(), all_distance_points.end(), g1, max_p, g2);

                //hipHostUnregister(reinterpret_cast<void *>(const_cast<uint8_t *>(cloud->data.data())));
                hipStreamSynchronize(stream);
                hipFree(gpu_data_cloud);
                hipStreamDestroy(stream);
                hipCtxSynchronize();
                hipCtxPopCurrent(&context);
                hipCtxDestroy(context);

                min_pt = min_value;
                max_pt = max_value;
                return true;
            }

            bool getMinMax3D (const pcl::PCLPointCloud2ConstPtr &cloud, int x_idx, int y_idx, int z_idx,
                         const std::string &distance_field_name, float min_distance, float max_distance,
                         Eigen::Vector4f &min_pt, Eigen::Vector4f &max_pt, bool limit_negative) {
        // @todo fix this
          if (cloud->fields[x_idx].datatype != pcl::PCLPointField::FLOAT32 ||
              cloud->fields[y_idx].datatype != pcl::PCLPointField::FLOAT32 ||
              cloud->fields[z_idx].datatype != pcl::PCLPointField::FLOAT32)
          {
            PCL_ERROR ("[pcl::getMinMax3D] XYZ dimensions are not float type!\n");
            return false;
          }

          Eigen::Array4f min_p, max_p;
          min_p.setConstant (FLT_MAX);
          max_p.setConstant (-FLT_MAX);

          // Get the distance field index
          int distance_idx = pcl::getFieldIndex (*cloud, distance_field_name);

          // @todo fix this
          if (cloud->fields[distance_idx].datatype != pcl::PCLPointField::FLOAT32)
          {
            PCL_ERROR ("[pcl::getMinMax3D] Filtering dimensions is not float type!\n");
            return false;
          }

          size_t nr_points = cloud->width * cloud->height;

          Eigen::Array4f pt = Eigen::Array4f::Zero ();


                hipInit(0);
                hipCtx_t context;
                hipDevice_t device;
                hipDeviceGet ( &device, 0 );
                hipCtxCreate(&context, hipDeviceScheduleBlockingSync | hipDeviceMapHost,device);
                hipCtxPushCurrent(context);
                hipStream_t stream;
                hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
                thrust::device_vector<Eigen::Array4f> all_distance_points(nr_points);


                uint8_t * gpu_data_cloud;
                hipMalloc(&gpu_data_cloud, sizeof(cloud->data[0]) * cloud->data.size());
                hipMemcpy(gpu_data_cloud, cloud->data.data(), sizeof(cloud->data[0]) * cloud->data.size(), hipMemcpyHostToDevice );

                //hipHostRegister(reinterpret_cast<void *>(const_cast<uint8_t *>(cloud->data.data())), sizeof(cloud->data[0]) * cloud->data.size(), hipHostRegisterPortable | hipHostRegisterMapped);
                //hipHostGetDevicePointer();

                //hipDeviceptr_t device_ptr;
                //hipHostGetDevicePointer(&device_ptr, reinterpret_cast<void *>(const_cast<uint8_t *>(cloud->data.data())), 0);
                //cuMemcpy();
                dim3 threadsPerBlock(1024, 1);
                dim3 numBlocks(static_cast<int> (std::ceil(static_cast<float>(cloud->data.size()) / threadsPerBlock.x)),
                               1);

                //kernel launch

                convertToDeviceVector<<<threadsPerBlock, numBlocks, 0, stream>>>((unsigned char *)gpu_data_cloud, nr_points, cloud->fields[x_idx].offset, cloud->fields[y_idx].offset, cloud->fields[z_idx].offset, cloud->point_step, thrust::raw_pointer_cast(&all_distance_points[0]));

                convert_indefinite_to_max_value_with_limits<Eigen::Array4f> f1(min_distance,max_distance,limit_negative,distance_idx, x_idx,y_idx,z_idx);
                compute_minimum_of_bounding_box<Eigen::Array4f> f2;
                convert_indefinite_to_min_value_with_limits<Eigen::Array4f> g1(min_distance,max_distance,limit_negative,distance_idx, x_idx,y_idx,z_idx);
                compute_maximum_of_bounding_box<Eigen::Array4f> g2;

                auto min_value = thrust::transform_reduce(thrust::cuda::par.on(stream), all_distance_points.begin(), all_distance_points.end(), f1, min_p, f2);
                auto max_value = thrust::transform_reduce(thrust::cuda::par.on(stream), all_distance_points.begin(), all_distance_points.end(), g1, max_p, g2);

                //hipHostUnregister(reinterpret_cast<void *>(const_cast<uint8_t *>(cloud->data.data())));
                hipStreamSynchronize(stream);
                hipFree(gpu_data_cloud);
                hipStreamDestroy(stream);
                hipCtxSynchronize();
                hipCtxPopCurrent(&context);
                hipCtxDestroy(context);

                min_pt = min_value;
                max_pt = max_value;
                return true;

            }
#endif
        }
    }
}